#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include "vec3.h"

__global__ void render(vec3 *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    fb[pixel_index] = vec3( float(i) / max_x, float(j) / max_y, 0.2f);
}

int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    // asignar "buffer" para los pixeles
    vec3 *fb;
    hipMallocManaged((void **)&fb, fb_size);

    clock_t start, stop;
    start = clock();
    // Calcular ("render") los valores de los pixeles 
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, nx, ny);
    hipGetLastError();
    hipDeviceSynchronize();
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Mandar el resultado al "standard out"
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    hipFree(fb);
}
