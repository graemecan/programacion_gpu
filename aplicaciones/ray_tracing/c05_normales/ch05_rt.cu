#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"

__device__ vec3 color(const ray& r, hitable **world) {
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5f*vec3(rec.normal.x()+1.0f, rec.normal.y()+1.0f, rec.normal.z()+1.0f);
    }
    else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f*(unit_direction.y() + 1.0f);
        return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
    }
}

__global__ void render(vec3 *fb, int max_x, int max_y,
                       vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin,
                       hitable **world) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    ray r(origin, lower_left_corner + u*horizontal + v*vertical);
    fb[pixel_index] = color(r, world);
}

__global__ void create_world(hitable **d_list, hitable **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list)   = new sphere(vec3(0,0,-1), 0.5);
        *(d_list+1) = new sphere(vec3(0,-100.5,-1), 100);
        *d_world    = new hitable_list(d_list,2);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world) {
    delete *(d_list);
    delete *(d_list+1);
    delete *d_world;
}

int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    // allocate FB
    vec3 *fb;
    hipMallocManaged((void **)&fb, fb_size);

    // make our world of hitables
    hitable **d_list;
    hipMalloc((void **)&d_list, 2*sizeof(hitable *));
    hitable **d_world;
    hipMalloc((void **)&d_world, sizeof(hitable *));
    create_world<<<1,1>>>(d_list,d_world);
    hipGetLastError();
    hipDeviceSynchronize();

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, nx, ny,
                                vec3(-2.0, -1.0, -1.0),
                                vec3(4.0, 0.0, 0.0),
                                vec3(0.0, 2.0, 0.0),
                                vec3(0.0, 0.0, 0.0),
                                d_world);
    hipGetLastError();
    hipDeviceSynchronize();
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    hipDeviceSynchronize();
    free_world<<<1,1>>>(d_list,d_world);
    hipGetLastError();
    hipFree(d_list);
    hipFree(d_world);
    hipFree(fb);

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}
