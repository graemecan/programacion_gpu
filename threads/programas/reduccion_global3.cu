
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void reduccion_memoria_global(float *data, int stride, int N) {

    unsigned long int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // para arrays muy grandes esta variable puede
    // ser tan grande que hay que usar long int
    unsigned long int index = 2 * stride * idx;
    if (index < N) {
        data[index] += data[index + stride];
    }
}

void inicializar_numeros(float *data, int size)
{
    for (int i = 0; i < size; i++)
    {
        // Generamos números aleatorios pequeños
        // El "0xFF" es un número en hexadecimal y está combinado con
        // rand() usando el "bitwise operator" & para obtener números
        // entre 0 y 255.
        data[i] = (float)(rand() & 0xFF) / (float)RAND_MAX;
    }
}

float resultado_cpu(float *data, int N)
{
    float resultado = 0.f;
    for (int i = 0; i < N; i++){
        resultado += data[i];
    }

    return resultado;
}

int main()
{
    float *h_array;
    float *d_array;

    unsigned int N = 1 << 24;
    unsigned int n_threads = 1024;

    float resultado_host, resultado_gpu;

    srand(2019);

    // Asignar memoria en el host
    h_array = (float *)malloc(N * sizeof(float));

    // Iniciliazar valores de h_array con números aleatorios
    inicializar_numeros(h_array, N);

    // Asignar memoria en el GPU y copiar datos
    hipMalloc((void **)&d_array, N * sizeof(float));
    hipMemcpy(d_array, h_array, N * sizeof(float), hipMemcpyHostToDevice);

    // Calcular reducción en el GPU
    int n_bloques = (N + n_threads - 1) / n_threads;
    for (int stride = 1; stride < N; stride *= 2) {
        reduccion_memoria_global<<<n_bloques, n_threads>>>(d_array, stride, N);
    }

    // Copiar resultado del GPU
    hipMemcpy(&resultado_gpu, &d_array[0], sizeof(float), hipMemcpyDeviceToHost);

    // Calcular reducción en el CPU (secuencial)
    resultado_host = resultado_cpu(h_array, N);
    printf("host: %f, device %f\n", resultado_host, resultado_gpu);

    // Liberar memoria
    hipFree(d_array);
    free(h_array);

    return 0;
}
