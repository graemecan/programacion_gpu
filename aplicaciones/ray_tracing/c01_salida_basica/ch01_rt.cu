
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

__global__ void render(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x*3 + i*3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}

int main() {
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    // mandamos mensajes a "standard error" manteniendo "standard out"
    // solamente para la salida de los datos de la imagen
    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = 3*num_pixels*sizeof(float);

    // asignar "buffer" (espacio en memoria para los pixeles)
    float *fb;
    hipMallocManaged((void **)&fb, fb_size);

    clock_t start, stop;
    start = clock();
    // Calcular ("render") los valores de los pixeles
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, nx, ny);
    hipGetLastError();
    hipDeviceSynchronize();
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Mandamos los datos de la imagen a "standard out"
    // Se puede redirigir "standard out" a un archivo con ">" en Linux
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*3*nx + i*3;
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.99*r);
            int ig = int(255.99*g);
            int ib = int(255.99*b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    hipFree(fb);
}
