
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

double seconds() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void mathKernel1(float *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;

    a = b = 0.0f;

    if (tid % 2 == 0) {
        a = 100.0f;
    } else {
        b = 200.0f;
    }
    c[tid] = a + b;
}

__global__ void mathKernel2(float *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;

    a = b = 0.0f;

    if ((tid / warpSize) % 2 == 0) {
        a = 100.0f;
    } else {
        b = 200.0f;
    }
    c[tid] = a + b;
}

__global__ void warmingup(float *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;

    a = b = 0.0f;

    if ((tid / warpSize) % 2 == 0) {
        a = 100.0f;
    } else {
        b = 200.0f;
    }
    c[tid] = a + b;
}

int main(int argc, char **argv) {
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    printf("%s using Device %d: %s\n", argv[0],dev, deviceProp.name);

    // set up data size
    int size = 64;
    int blocksize = 64;

    if(argc > 1) blocksize = atoi(argv[1]);
    if(argc > 2) size = atoi(argv[2]);

    printf("Data size %d ", size);

    // set up execution configuration
    dim3 block (blocksize,1);
    dim3 grid ((size+block.x-1)/block.x,1);
    printf("Execution Configure (block %d grid %d)\n",block.x, grid.x);

    // allocate gpu memory
    float *d_C;
    size_t nBytes = size * sizeof(float);
    hipMalloc((float**)&d_C, nBytes);

    // run a warmup kernel to remove overhead
    double iStart,iElaps;
    hipDeviceSynchronize();
    iStart = seconds();
    warmingup<<<grid, block>>> (d_C);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    printf("warmingup <<< %4d %4d >>> elapsed %f sec \n",grid.x,block.x, iElaps );

    // run kernel 1
    iStart = seconds();
    mathKernel1<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    printf("mathKernel1 <<< %4d %4d >>> elapsed %f sec \n",grid.x,block.x,iElaps );

    // run kernel 2
    iStart = seconds();
    mathKernel2<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    iElaps = seconds () - iStart;
    printf("mathKernel2 <<< %4d %4d >>> elapsed %f sec \n",grid.x,block.x,iElaps );

    // run kernel 3
    /*
    iStart = seconds ();
    mathKernel3<<<grid, block>>>(d_C);
    cudaDeviceSynchronize();
    iElaps = seconds () - iStart;
    printf("mathKernel3 <<< %4d %4d >>> elapsed %d sec \n",grid.x,block.x,iElaps);

    // run kernel 4
    iStart = seconds ();
    mathKernel4<<<grid, block>>>(d_C);
    cudaDeviceSynchronize();
    iElaps = seconds () - iStart;
    printf("mathKernel4 <<< %4d %4d >>> elapsed %d sec \n",grid.x,block.x,iElaps);
    */

    // free gpu memory and reset divece
    hipFree(d_C);
    hipDeviceReset();

    return EXIT_SUCCESS;
}
